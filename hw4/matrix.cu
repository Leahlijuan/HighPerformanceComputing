#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include "utils.h"

#define BLOCK_SIZE 1024

void Check_CUDA_Error(const char *message)
{
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
  {
    fprintf(stderr, "ERROR: %s: %s\n", message, hipGetErrorString(error));
    exit(-1);
  }
}


void matrix_vector_ref(double *Ax_ref, const double *A, const double *x, long N)
{
  for (long i = 0; i < N; i++)
  {
    for (long j = 0; j < N; j++)
    {
      Ax_ref[i] += A[N * i + j] * x[j];
    }
  }
}

__global__ void matrix_vector_kernel(double *Ax, const double *A, const double *x, long N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N)
  {
    Ax[idx] = 0;
    for (long j = 0; j < N; j++)
    {
      Ax[idx] += A[idx * N + j] * x[j];
    }
  }
}

int main(int argc, char **argv)
{
  long N = read_option<long>("-n", argc, argv);

  double *A, *x, *A_d, *x_d, *Ax_d, *Ax_ref, *Ax;

  // Initialize vector and matrix
  hipHostMalloc((void **)&A, N * N * sizeof(double));
  hipHostMalloc((void **)&x, N * sizeof(double));

  for (long i = 0; i < N; i++)
  {
    x[i] = drand48();
  }
  for (long i = 0; i < N * N; i++)
  {
    A[i] = drand48();
  }

  // Get reference product
  hipHostMalloc((void **)&Ax_ref, N * sizeof(double));
  Timer t;
  t.tic();
  matrix_vector_ref(Ax_ref, A, x, N);
  double time = t.toc();
  printf("CPU Bandwidth = %f GB/s\n", 2 * N * N * sizeof(double) / time / 1e9);

  // Get GPU product
  hipMalloc(&A_d, N * N * sizeof(double));
  hipMalloc(&x_d, N * sizeof(double));
  hipMalloc(&Ax_d, N * sizeof(double));
  hipHostMalloc(&Ax, N * sizeof(double));

  hipMemcpyAsync(A_d, A, N * N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(x_d, x, N * sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  Timer t2;
  t2.tic();
  matrix_vector_kernel<<<N / BLOCK_SIZE + 1, BLOCK_SIZE>>>(Ax_d, A_d, x_d, N);
  hipMemcpyAsync(Ax, Ax_d, N * sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  double time2 = t2.toc();
  printf("GPU Bandwidth = %f GB/s\n", 2 * N * N * sizeof(double) / time2 / 1e9);

  double err = 0;
  for (long i = 0; i < N; i++)
  {
    err += (Ax_ref[i] - Ax[i]) * (Ax_ref[i] - Ax[i]);
  }
  printf("Error = %f\n", err);

  // Cleanup
  hipFree(A_d);
  hipFree(x_d);
  hipFree(Ax_d);
  hipHostFree(A);
  hipHostFree(x);
  hipHostFree(Ax);
  hipHostFree(Ax_ref);

  return 0;
}