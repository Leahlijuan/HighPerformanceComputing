#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils.h"
#include <math.h>
#include <stdlib.h>

double sum_of_neighbors(int N, double *u, int i, int j)
{
    double val_left = 0.0, val_right = 0.0, val_up = 0.0, val_down = 0.0;

    if (i > 0)
    {
        val_up = u[(i - 1) * N + j];
    }
    if (i < N - 1)
    {
        val_down = u[(i + 1) * N + j];
    }
    if (j > 0)
    {
        val_left = u[i * N + (j - 1)];
    }
    if (j < N - 1)
    {
        val_right = u[i * N + (j + 1)];
    }

    return val_left + val_right + val_up + val_down;
}

void A_times(int N, double *u, double *ret)
{
    double h_sq = 1.0 / ((N + 1) * (N + 1));

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int flat_ix = i * N + j;

            double val_here = u[flat_ix];

            ret[flat_ix] =
                1.0 / h_sq *
                (4.0 * val_here - sum_of_neighbors(N, u, i, j));
        }
    }
}

double residual(int N, double *u, double *f)
{
    double *Au = (double *)malloc(N * N * sizeof(double));
    A_times(N, u, Au);
    double residual_norm_sq = 0.0;
    for (int i = 0; i < N * N; i++)
    {
        residual_norm_sq += (Au[i] - f[i]) * (Au[i] - f[i]);
    }

    free(Au);

    return sqrt(residual_norm_sq);
}

void jacobi_iters_par(int N, int iters, double *f, double *u_prev, double *u)
{
    double h_sq = 1.0 / ((N + 1) * (N + 1));

    for (int ix = 0; ix < iters; ix++)
    {

        for (int i = 0; i < N; i++)
        {
            for (int j = 0; j < N; j++)
            {
                int flat_ix = i * N + j;
                u[flat_ix] = 0.25 *
                             (h_sq * f[flat_ix] + sum_of_neighbors(N, u_prev, i, j));
            }
        }

        // Swap u and u_prev
        double *tmp = u;
        u = u_prev;
        u_prev = tmp;
    }

    // swap
    double *tmp = u;
    u = u_prev;
    u_prev = tmp;
}

#define BLOCK_SIZE 32

__global__ void jacobi_kernel(double *u, const double *f, const double *u_prev, double h_sq, long N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    double val_left = 0.0, val_right = 0.0, val_up = 0.0, val_down = 0.0;

    if (i < N && j < N)
    {
        if (i > 0)
        {
            val_up = u_prev[(i - 1) * N + j];
        }
        if (i < N - 1)
        {
            val_down = u_prev[(i + 1) * N + j];
        }
        if (j > 0)
        {
            val_left = u_prev[i * N + (j - 1)];
        }
        if (j < N - 1)
        {
            val_right = u_prev[i * N + (j + 1)];
        }

        u[N * i + j] = 0.25 * (h_sq * f[N * i + j] + val_up + val_down + val_left + val_right);
    }
}

void jacobi_wrapper(double *u, const double *f, long N, long iters)
{
    double *u_d, *f_d, *u_prev_d;

    hipMalloc(&u_d, N * N * sizeof(double));
    hipMalloc(&f_d, N * N * sizeof(double));
    hipMalloc(&u_prev_d, N * N * sizeof(double));

    // Initialize both current and previous step to input
    hipMemcpyAsync(u_d, u, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(u_prev_d, u, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(f_d, f, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    double h_sq = 1.0 / ((N + 1) * (N + 1));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(N / BLOCK_SIZE + 1, N / BLOCK_SIZE + 1);

    // Iterate kernel, copying current step to previous each time
    for (int k = 0; k < iters; k++)
    {
        jacobi_kernel<<<dimGrid, dimBlock>>>(u_d, f_d, u_prev_d, h_sq, N);
        hipMemcpyAsync(u_prev_d, u_d, N * N * sizeof(double), hipMemcpyDeviceToDevice);
        hipDeviceSynchronize();
    }

    // Return final result to host memory
    hipMemcpyAsync(u, u_prev_d, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(u_d);
    hipFree(f_d);
    hipFree(u_prev_d);
}

int main(int argc, char **argv)
{
    // General setup
    long N = read_option<long>("-n", argc, argv);
    long iters = 5000;

    Timer timer;

    double *f = (double *)malloc(N * N * sizeof(double));
    for (int i = 0; i < N * N; i++)
    {
        f[i] = 1.0;
    }

    // CPU calculation
    double *u_prev = (double *)malloc(N * N * sizeof(double));
    double *u = (double *)malloc(N * N * sizeof(double));
    for (int i = 0; i < N * N; i++)
    {
        u_prev[i] = 0.0;
        u[i] = 0.0;
    }

    printf("CPU:\n");
    timer.tic();
    jacobi_iters_par(N, iters, f, u, u_prev);

    double time = timer.toc();

    printf("residual:   %3f\n", residual(N, u, f));
    printf("time:   %3f\n\n", time);

    free(u_prev);
    free(u);

    // GPU calculation
    hipHostMalloc(&u, N * N * sizeof(double));
    for (int i = 0; i < N * N; i++)
    {
        u[i] = 0.0;
    }

    printf("GPU:\n");

    timer.tic();
    jacobi_wrapper(u, f, N, iters);
    time = timer.toc();

    printf("residual:   %3f\n", residual(N, u, f));
    printf("time:   %3f\n\n", time);

    free(f);
    hipFree(u);

    return 0;
}